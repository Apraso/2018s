
#include <hip/hip_runtime.h>
__global__ void add_one(int *x) {
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  x[index] = x[index] + 1;
}

int main() {
  int x[256];
  int* x_gpu;
  hipMalloc(&x_gpu, 256 * sizeof(int));
  hipMemcpy(x_gpu, x, 256 * sizeof(int), hipMemcpyHostToDevice);

  add_one<<<1, 256>>>(x_gpu);
}
